#include <hip/hip_runtime.h>
#include <iostream>

int main() {
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);

    std::cout << "Device Name: " << prop.name << std::endl;
    std::cout << "Max Threads per Block: " << prop.maxThreadsPerBlock << std::endl;
    std::cout << "Max Threads per SM: " << prop.maxThreadsPerMultiProcessor << std::endl;
    std::cout << "Max Grid Size: (" << prop.maxGridSize[0] << ", " << prop.maxGridSize[1] << ", " << prop.maxGridSize[2] << ")" << std::endl;
    std::cout << "Shared Memory per Block: " << prop.sharedMemPerBlock << " bytes" << std::endl;
    std::cout << "Total Global Memory: " << prop.totalGlobalMem / (1024 * 1024) << " MB" << std::endl;
    std::cout << "Registers per Block: " << prop.regsPerBlock << std::endl;
    std::cout << "Warp Size: " << prop.warpSize << std::endl;

    return 0;
}