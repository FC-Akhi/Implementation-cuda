// This case works only for square matrix


#include <hip/hip_runtime.h>
#include <iostream>
#include <stdint.h>

unsigned int *generator(unsigned int img_width, unsigned int img_height, unsigned int tile_size, unsigned int *index_arr)
{
    
    unsigned int block_size = tile_size * tile_size;


    int index = 0;

    for (unsigned int i = 0; i < (img_width * img_height); i++){

        
        index = ((i % tile_size) + 
                (((i / tile_size) % (img_width  / tile_size)) * block_size) + 
                ((i / img_width) * tile_size) + 
                ((i / (img_width * tile_size))  * block_size));

        index_arr[i] = index;
        printf("i: %d, index: %d\n", i, index);

    }


    return index_arr;
}

void unit_testing(unsigned int img_width, unsigned int img_height, unsigned int tile_size, unsigned int *index_arr)
{

    unsigned int index_pal = 0;
    unsigned int block_size = tile_size * tile_size;

    
    for (unsigned int i = 0; i < (img_width * img_height); i++)
    {

        index_pal = (
            + ( (i/(img_width * tile_size)) * (img_width * tile_size) )
            + (i % tile_size)
            + ( ( (i / tile_size) % (img_width / tile_size) ) * block_size)
            + ( ((i/img_width) % tile_size) * tile_size)
        );

        
        printf("i: %d, index: %d\n", i, index_pal);
        
        
        if (index_pal != index_arr[i]){
            printf("Mismatched for %d\n", i);
            printf("i: %d, index_pal: %d\n", i, index_pal);
        }
        

    }
}



void initialData(float *ip,int size) {
    
    // generate different seed for random number
    time_t t;

    srand((unsigned) time(&t));
    
    for (int i=0; i<size; i++) {
    
        ip[i] = (float)( rand() & 0xFF )/10.0f;
    
    }

}

__global__ void offseting_GPU(float *A, const int nx, const int N) {

    // int i = threadIdx.x;
    int ix = blockIdx.x * blockDim.x + threadIdx.x;
    int iy = blockIdx.y * blockDim.y + threadIdx.y;

    int idx = iy * nx + ix;


    if (idx < N)
        


        printf("threadIdx:(%d, %d, %d) blockIdx:(%d, %d, %d) A[%d]\n", threadIdx.x, threadIdx.y, threadIdx.z,
        blockIdx.x, blockIdx.y, blockIdx.z, idx);


}


int main(void) {

    int nx = 4;
    int ny = 4;

    int nxy = nx*ny;
    int nBytes = nxy * sizeof(float);


    // malloc host global memory
    float *h_A;

    h_A = (float *)malloc (nBytes);
    

    // initialize data at host side
    initialData(h_A, nxy);


    // malloc device global memory
    float *d_A;
    hipMalloc((float**)&d_A, nBytes);



    // transfer data from host to device
    hipMemcpy(d_A, h_A, nBytes, hipMemcpyHostToDevice);



    // invoke kernel at host side
    dim3 block (nx / 2, ny / 2);
    dim3 grid (block.x, block.y);

    printf("Execution configuration <<<%d, %d, %d, %d>>>\n", grid.x, grid.y, block.x, block.y);


    // double iStart = cpuSecond();
    offseting_GPU<<< grid, block >>>(d_A, nx, nxy);
    hipDeviceSynchronize();
    // double iElaps = cpuSecond() - iStart;

    
    





    // // check device results
    // checkResult(hostRef, gpuRef, nElem);
    
    
    // free device global memory
    hipFree(d_A);

    
    
    // free host memory
    free(h_A);

    
    return(0);

}